#include <stdio.h>
#include <chrono>

#define TESTS 10
#define N 1024 * 1024 * 1
#define BLOCK_SIZE 256
#define GRID_SIZE 1024

#include "step.hu"

float randf()
{
  return (float)((double)rand() / RAND_MAX);
}

int main()
{
  statDevice();

  // Screen limits
  float width = 1920;
  float height = 1080;

  printf("Expected Memory usage: %lu MB\n", (sizeof(float) * N * 6) / 1024 / 1024);

  // Allocate px and py in host memory
  float *h_px, *h_py;
  checkCudaErrors(hipHostMalloc(&h_px, sizeof(float) * N));
  checkCudaErrors(hipHostMalloc(&h_py, sizeof(float) * N));

  // Fill with random values in the screen
  for (int i = 0; i < N; i++)
  {
    h_px[i] = randf() * width;
    h_py[i] = randf() * height;
  }

  // Allocate vectors in device memory
  float *d_px, *d_py, *d_vx, *d_vy, *d_fx, *d_fy;
  checkCudaErrors(hipMalloc(&d_px, sizeof(float) * N));
  checkCudaErrors(hipMalloc(&d_py, sizeof(float) * N));
  checkCudaErrors(hipMalloc(&d_vx, sizeof(float) * N));
  checkCudaErrors(hipMalloc(&d_vy, sizeof(float) * N));
  checkCudaErrors(hipMalloc(&d_fx, sizeof(float) * N));
  checkCudaErrors(hipMalloc(&d_fy, sizeof(float) * N));

  // Copy host positions to device
  checkCudaErrors(hipMemcpy(d_px, h_px, sizeof(float) * N, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_py, h_py, sizeof(float) * N, hipMemcpyHostToDevice));

  // Vectors can be initialized with zeros
  checkCudaErrors(hipMemset(d_vx, 0, sizeof(float) * N));
  checkCudaErrors(hipMemset(d_vy, 0, sizeof(float) * N));

  // Force vectors do not need to be initialized

  hipStream_t compute_stream;
  // Allocate the stream
  checkCudaErrors(hipStreamCreate(&compute_stream));

  auto start = std::chrono::high_resolution_clock::now();

  // Launch the kernel
  for (int i = 0; i < TESTS; i++)
  {
    step<BLOCK_SIZE, GRID_SIZE>(compute_stream, d_px, d_py, d_vx, d_vy, d_fx, d_fy);
  }

  // Wait for the stream to finish
  checkCudaErrors(hipStreamSynchronize(compute_stream));

  auto end = std::chrono::high_resolution_clock::now();

  // Print the time
  printf("Average time (ms): %lf\n", (double)std::chrono::duration_cast<std::chrono::nanoseconds>((end - start) / TESTS).count() / 1000000.);
  printf("Expected FPS: %lf\n", 1000000000. / (double)std::chrono::duration_cast<std::chrono::nanoseconds>((end - start) / TESTS).count());

  // Free memory
  checkCudaErrors(hipFree(d_px));
  checkCudaErrors(hipFree(d_py));
  checkCudaErrors(hipFree(d_vx));
  checkCudaErrors(hipFree(d_vy));
  checkCudaErrors(hipFree(d_fx));
  checkCudaErrors(hipFree(d_fy));
}