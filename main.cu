#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <cuda_gl_interop.h>
#include "helper_cuda.cuh"

#define N 108 * 512 * 6
#define BLOCK_SIZE 512
#define GRID_SIZE 108
#define GRAVITY 1.f
#define SOFTENING 0.001f
#define DELTA_T 0.01f

#include "step.cuh"

unsigned int window_width = 2560,
             window_height = 1400;

float height = 500.;
float width = 500.;
float2 center = {0, 0};

bool mouse1_down = false;

double mouse_x, mouse_y;

float randf()
{
  return (float)((double)rand() / RAND_MAX);
}

void update_view()
{
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  glOrtho(center.x - width / 2, center.x + width / 2, center.y - height / 2, center.y + height / 2, -1, 1);
}

void resize(GLFWwindow *window, int new_window_width, int new_window_height)
{
  window_width = new_window_width;
  window_height = new_window_height;

  // Adjust width to the aspect ratio
  width = height * (float)window_width / (float)window_height;

  glViewport(0, 0, window_width, window_height);
  update_view();
}

void input_handler(GLFWwindow *window, int key, int scancode, int action, int mods)
{
  if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
    glfwSetWindowShouldClose(window, GLFW_TRUE);

  if (key == GLFW_KEY_W && action != GLFW_RELEASE)
  {
    center.y += height / 50.;
    update_view();
  }
  if (key == GLFW_KEY_S && action != GLFW_RELEASE)
  {
    center.y -= height / 50.;
    update_view();
  }
  if (key == GLFW_KEY_A && action != GLFW_RELEASE)
  {
    center.x -= height / 50.;
    update_view();
  }
  if (key == GLFW_KEY_D && action != GLFW_RELEASE)
  {
    center.x += height / 50.;
    update_view();
  }
}

void scroll_handler(GLFWwindow *window, double xoffset, double yoffset)
{
  double normalized_x = 2.f * mouse_x / window_width - 1.f;
  double normalized_y = 1.f - 2.f * mouse_y / window_height;

  // get the position of the mouse in the world coordinates
  float xpos = normalized_x * width / 2 + center.x;
  float ypos = normalized_y * height / 2 + center.y;

  // get the new width
  width *= 1 + yoffset / 10.;
  height *= 1 + yoffset / 10.;

  // adjust the center to keep the mouse position fixed
  center.x = xpos - normalized_x * width / 2;
  center.y = ypos - normalized_y * height / 2;
  update_view();
}

void mouse_handler(GLFWwindow *window, int button, int action, int mods)
{
  if (button == GLFW_MOUSE_BUTTON_1 && action == GLFW_PRESS)
  {
    mouse1_down = true;
  }
  if (button == GLFW_MOUSE_BUTTON_1 && action == GLFW_RELEASE)
  {
    mouse1_down = false;
  }
}

void pan_handler(GLFWwindow *window, double xpos, double ypos)
{
  if (mouse1_down)
  {
    center.x -= (xpos - mouse_x) * width / window_width;
    center.y += (ypos - mouse_y) * height / window_height;
    update_view();
  }

  mouse_x = xpos;
  mouse_y = ypos;
}

int main()
{
  unsigned int window_width = 2560,
               window_height = 1400;

  float start_radius = 100.0f;

  GLFWwindow *window;

  /* Initialize the library */
  if (!glfwInit())
    return -1;

  /* Create a windowed mode window and its OpenGL context */
  char window_name[100];
  sprintf(window_name, "Particle simulation with %d particles", N);
  window = glfwCreateWindow(window_width, window_height, window_name, NULL, NULL);
  if (!window)
  {
    glfwTerminate();
    return -1;
  }

  /* Make the window's context current */
  glfwMakeContextCurrent(window);

  resize(window, window_width, window_height);
  glfwSetFramebufferSizeCallback(window, resize);

  /// Set up callbacks
  glfwSetKeyCallback(window, input_handler);
  glfwSetScrollCallback(window, scroll_handler);
  glfwSetMouseButtonCallback(window, mouse_handler);
  glfwSetCursorPosCallback(window, pan_handler);

  hipSetDevice(0);

  statDevice();

  // Allocate px and py in host memory
  float2 *h_p, *h_v;
  float2 *d_p, *d_v, *d_f;
  checkCudaErrors(hipHostMalloc(&h_p, sizeof(float2) * N));
  checkCudaErrors(hipHostMalloc(&h_v, sizeof(float2) * N));

  // Fill with random values in the screen
  for (int i = 0; i < N; i++)
  {
    float theta = randf() * 2 * M_PI;
    float distance = randf() * start_radius;

    h_p[i].x = distance * cos(theta);
    h_p[i].y = distance * sin(theta);

    // Random velocity
    h_v[i].x = (randf() * 2 - 1) * start_radius / 10;
    h_v[i].y = (randf() * 2 - 1) * start_radius / 10;
  }

  printf("Initialized %d particles\n", N);

  if (GLenum err = glewInit() != GLEW_OK)
  {
    printf("Failed to initialize GLEW: error code %u\n", err);
    return -1;
  }

  size_t positions_size = N * sizeof(float2);

  GLuint positions;
  glGenBuffers(1, &positions);
  // Copy to the openGL buffer
  glBindBuffer(GL_ARRAY_BUFFER, positions);
  glBufferData(GL_ARRAY_BUFFER, positions_size, h_p, GL_DYNAMIC_DRAW);
  glBindBuffer(GL_ARRAY_BUFFER, 0);

#ifdef __WSL__
  // WSL does not support OpenGL interoperability
  checkCudaErrors(hipMalloc(&d_p, positions_size));
  checkCudaErrors(hipMemcpy(d_p, h_p, positions_size, hipMemcpyHostToDevice));
#else
  // Create buffer object and register it with CUDA
  hipGraphicsResource_t positions_resource;

  // Register the buffer with CUDA
  checkCudaErrors(hipGraphicsGLRegisterBuffer(&positions_resource, positions, hipGraphicsRegisterFlagsNone));
#endif

  // Allocate vectors in device memory
  checkCudaErrors(hipMalloc(&d_v, positions_size));
  checkCudaErrors(hipMalloc(&d_f, positions_size));

  // Copy velocities to the device
  checkCudaErrors(hipMemcpy(d_v, h_v, positions_size, hipMemcpyHostToDevice));

  // Force vectors do not need to be initialized

  hipStream_t compute_stream;
  // Allocate the stream
  checkCudaErrors(hipStreamCreate(&compute_stream));

  glColor3f(1.0f, 1.0f, 1.0f);
  glPointSize(1.0f);

  /* Loop until the user closes the window */
  while (!glfwWindowShouldClose(window))
  {
    /* Render here */
    glClear(GL_COLOR_BUFFER_BIT);

    // Draw the particles
    glBindBuffer(GL_ARRAY_BUFFER, positions);
    glVertexPointer(2, GL_FLOAT, 0, 0);
    glEnableClientState(GL_VERTEX_ARRAY);
    glDrawArrays(GL_POINTS, 0, N);
    glDisableClientState(GL_VERTEX_ARRAY);

    /* Swap front and back buffers */
    glfwSwapBuffers(window);

    /* Poll for and process events */
    glfwPollEvents();

#ifdef __WSL__
    // Launch the kernel
    step<BLOCK_SIZE, GRID_SIZE>(compute_stream, d_p, d_v, d_f);
    // Wait for the kernel to finish
    checkCudaErrors(hipStreamSynchronize(compute_stream));

    // Copy the results back to the host
    checkCudaErrors(hipMemcpy(h_p, d_p, positions_size, hipMemcpyDeviceToHost));

    // Copy the results to the OpenGL buffer
    glBindBuffer(GL_ARRAY_BUFFER, positions);
    glBufferData(GL_ARRAY_BUFFER, positions_size, h_p, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);

#else
    // Launch the kernel
    checkCudaErrors(hipGraphicsMapResources(1, &positions_resource, compute_stream));
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&d_p, &positions_size, positions_resource));

    step<BLOCK_SIZE, GRID_SIZE>(compute_stream, d_p, d_v, d_f);

    // Wait for the kernel to finish
    checkCudaErrors(hipStreamSynchronize(compute_stream));
    hipGraphicsUnmapResources(1, &positions_resource, compute_stream);
#endif
  }

  glfwTerminate();

  checkCudaErrors(hipFree(d_v));
  checkCudaErrors(hipFree(d_f));
  return 0;
}