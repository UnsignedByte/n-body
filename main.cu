#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <cuda_gl_interop.h>
#include "helper_cuda.hu"

#define N 108 * 512 * 6
#define BLOCK_SIZE 512
#define GRID_SIZE 108
#define GRAVITY 1.f
#define SOFTENING 0.001f
#define DELTA_T 0.01f

#include "step.hu"

float randf()
{
  return (float)((double)rand() / RAND_MAX);
}

void resize(GLFWwindow *window, int width, int height)
{
  glViewport(0, 0, width, height);
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  glOrtho(0, width, height, 0, -1, 1);
}

int main()
{
  unsigned int width = 2560,
               height = 1400;

  GLFWwindow *window;

  /* Initialize the library */
  if (!glfwInit())
    return -1;

  /* Create a windowed mode window and its OpenGL context */
  char window_name[100];
  sprintf(window_name, "Particle simulation with %d particles", N);
  window = glfwCreateWindow(width, height, window_name, NULL, NULL);
  if (!window)
  {
    glfwTerminate();
    return -1;
  }

  /* Make the window's context current */
  glfwMakeContextCurrent(window);

  resize(window, width, height);
  glfwSetFramebufferSizeCallback(window, resize);

  hipSetDevice(0);

  statDevice();

  // Allocate px and py in host memory
  float2 *h_p;
  float2 *d_p, *d_v, *d_f;
  checkCudaErrors(hipHostMalloc(&h_p, sizeof(float2) * N));

  // Fill with random values in the screen
  for (int i = 0; i < N; i++)
  {
    h_p[i].x = randf() * width;
    h_p[i].y = randf() * height;
  }

  printf("Initialized %d particles\n", N);

  if (GLenum err = glewInit() != GLEW_OK)
  {
    printf("Failed to initialize GLEW: error code %u\n", err);
    return -1;
  }

  size_t positions_size = N * sizeof(float2);

  GLuint positions;
  glGenBuffers(1, &positions);
  // Copy to the openGL buffer
  glBindBuffer(GL_ARRAY_BUFFER, positions);
  glBufferData(GL_ARRAY_BUFFER, positions_size, h_p, GL_DYNAMIC_DRAW);
  glBindBuffer(GL_ARRAY_BUFFER, 0);

#ifdef __WSL__
  // WSL does not support OpenGL interoperability
  checkCudaErrors(hipMalloc(&d_p, positions_size));
  checkCudaErrors(hipMemcpy(d_p, h_p, positions_size, hipMemcpyHostToDevice));
#else
  // Create buffer object and register it with CUDA
  hipGraphicsResource_t positions_resource;

  // Register the buffer with CUDA
  checkCudaErrors(hipGraphicsGLRegisterBuffer(&positions_resource, positions, hipGraphicsRegisterFlagsNone));
#endif

  // Allocate vectors in device memory
  checkCudaErrors(hipMalloc(&d_v, positions_size));
  checkCudaErrors(hipMalloc(&d_f, positions_size));

  // Vectors must be initialized with zeros
  checkCudaErrors(hipMemset(d_v, 0, positions_size));
  // Force vectors do not need to be initialized

  hipStream_t compute_stream;
  // Allocate the stream
  checkCudaErrors(hipStreamCreate(&compute_stream));

  glColor3f(1.0f, 1.0f, 1.0f);
  glPointSize(1.0f);

  /* Loop until the user closes the window */
  while (!glfwWindowShouldClose(window))
  {
    /* Render here */
    glClear(GL_COLOR_BUFFER_BIT);

    // Draw the particles
    glBindBuffer(GL_ARRAY_BUFFER, positions);
    glVertexPointer(2, GL_FLOAT, 0, 0);
    glEnableClientState(GL_VERTEX_ARRAY);
    glDrawArrays(GL_POINTS, 0, N);
    glDisableClientState(GL_VERTEX_ARRAY);

    /* Swap front and back buffers */
    glfwSwapBuffers(window);

    /* Poll for and process events */
    glfwPollEvents();

#ifdef __WSL__
    // Launch the kernel
    step<BLOCK_SIZE, GRID_SIZE>(compute_stream, d_p, d_v, d_f);
    // Wait for the kernel to finish
    checkCudaErrors(hipStreamSynchronize(compute_stream));

    // Copy the results back to the host
    checkCudaErrors(hipMemcpy(h_p, d_p, positions_size, hipMemcpyDeviceToHost));

    // Copy the results to the OpenGL buffer
    glBindBuffer(GL_ARRAY_BUFFER, positions);
    glBufferData(GL_ARRAY_BUFFER, positions_size, h_p, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);

#else
    // Launch the kernel
    checkCudaErrors(hipGraphicsMapResources(1, &positions_resource, compute_stream));
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&d_p, &positions_size, positions_resource));

    step<BLOCK_SIZE, GRID_SIZE>(compute_stream, d_p, d_v, d_f);

    // Wait for the kernel to finish
    checkCudaErrors(hipStreamSynchronize(compute_stream));
    hipGraphicsUnmapResources(1, &positions_resource, compute_stream);
#endif
  }

  glfwTerminate();

  checkCudaErrors(hipFree(d_v));
  checkCudaErrors(hipFree(d_f));
  return 0;
}